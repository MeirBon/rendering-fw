#include "hip/hip_runtime.h"
#include "Shared.h"

#include <hip/hip_runtime.h>
#include <>

#include "BVH/BVHNode.h"
#include "BVH/MBVHNode.h"

#include "CUDAIntersect.h"

#include "getShadingData.h"

#include "bsdf.h"
#include "lights.h"

#define USE_TOP_MBVH 1
#define USE_MBVH 1
#define IS_SPECULAR 1
#define IS_LIGHTS 1
#define MAX_IS_LIGHTS 16

#define NEXTMULTIPLEOF(a, b) (((a) + ((b)-1)) & (0x7fffffff - ((b)-1)))
using namespace glm;

#ifndef __launch_bounds__ // Fix errors in IDE
void __sincosf(float, float *, float *) {}

#define __launch_bounds__(x, y)
int __float_as_int(float x) { return int(x); }
uint __float_as_uint(float x) { return uint(x); }
float __uint_as_float(uint x) { return float(x); }
float __int_as_float(int x) { return float(x); }

template <typename T, typename B> T atomicAdd(T *, B) { return T; }

template <typename T, int x> struct surface
{
};
template <typename T> void surf2Dwrite(T value, surface<void, cudaSurfaceType2D> output, size_t stride, size_t y, hipSurfaceBoundaryMode mode) {}
#endif

surface<void, cudaSurfaceType2D> output;

__constant__ __device__ float geometryEpsilon;
__constant__ __device__ CameraView *view;
__constant__ __device__ Counters *counters;
__constant__ __device__ glm::vec4 *accumulator;
__constant__ __device__ uint stride;
__constant__ __device__ glm::vec4 *pathStates;
__constant__ __device__ glm::vec4 *pathOrigins;
__constant__ __device__ glm::vec4 *pathDirections;
__constant__ __device__ glm::vec4 *pathThroughputs;
__constant__ __device__ glm::vec3 *skybox;
__constant__ __device__ uint skyboxWidth;
__constant__ __device__ uint skyboxHeight;
__constant__ __device__ uint scrWidth;
__constant__ __device__ uint scrHeight;
__constant__ __device__ uint *blueNoise;
__constant__ __device__ float clampValue;

__constant__ __device__ PotentialContribution *connectData;
__constant__ __device__ DeviceInstanceDescriptor *instances;

#ifndef MAT_CONSTANTS_H
#define MAT_CONSTANTS_H
__constant__ __device__ DeviceMaterial *materials;
__constant__ __device__ glm::vec4 *floatTextures;
__constant__ __device__ uint *uintTextures;
#endif

#ifndef LIGHTS_H
#define LIGHTS_H
__constant__ __device__ rfw::DeviceAreaLight *areaLights;
__constant__ __device__ rfw::DevicePointLight *pointLights;
__constant__ __device__ rfw::DeviceSpotLight *spotLights;
__constant__ __device__ rfw::DeviceDirectionalLight *directionalLights;
__constant__ __device__ rfw::LightCount lightCounts;
#endif

__constant__ __device__ glm::mat4 *instance_transforms;
__constant__ __device__ glm::mat4 *inverse_transforms;

__constant__ __device__ rfw::bvh::BVHNode *topLevelBVH;
__constant__ __device__ rfw::bvh::MBVHNode *topLevelMBVH;
__constant__ __device__ uint *topPrimIndices;
__constant__ __device__ rfw::bvh::AABB *topAABBs;

__constant__ __device__ rfw::bvh::BVHNode **meshBVHs;
__constant__ __device__ rfw::bvh::MBVHNode **meshMBVHs;
__constant__ __device__ uint **meshPrimIndices;
__constant__ __device__ glm::vec4 **meshVertices;
__constant__ __device__ glm::uvec3 **meshIndices;

__host__ void setTopLevelBVH(rfw::bvh::BVHNode *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(topLevelBVH), &ptr, sizeof(void *)); }
__host__ void setTopLevelMBVH(rfw::bvh::MBVHNode *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(topLevelMBVH), &ptr, sizeof(void *)); }
__host__ void setTopPrimIndices(uint *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(topPrimIndices), &ptr, sizeof(void *)); }
__host__ void setTopAABBs(rfw::bvh::AABB *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(topAABBs), &ptr, sizeof(void *)); }

__host__ void setInstanceTransforms(glm::mat4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(instance_transforms), &ptr, sizeof(void *)); }
__host__ void setInverseTransforms(glm::mat4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(inverse_transforms), &ptr, sizeof(void *)); }

__host__ void setMeshBVHs(rfw::bvh::BVHNode **ptr) { hipMemcpyToSymbol(HIP_SYMBOL(meshBVHs), &ptr, sizeof(void *)); }
__host__ void setMeshMBVHs(rfw::bvh::MBVHNode **ptr) { hipMemcpyToSymbol(HIP_SYMBOL(meshMBVHs), &ptr, sizeof(void *)); }
__host__ void setMeshBVHPrimIDs(uint **ptr) { hipMemcpyToSymbol(HIP_SYMBOL(meshPrimIndices), &ptr, sizeof(void *)); }
__host__ void setMeshVertices(glm::vec4 **ptr) { hipMemcpyToSymbol(HIP_SYMBOL(meshVertices), &ptr, sizeof(void *)); }
__host__ void setMeshIndices(glm::uvec3 **ptr) { hipMemcpyToSymbol(HIP_SYMBOL(meshIndices), &ptr, sizeof(void *)); }

__host__ void setCameraView(rfw::CameraView *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(view), &ptr, sizeof(void *)); }
__host__ void setCounters(Counters *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(counters), &ptr, sizeof(void *)); }
__host__ void setAccumulator(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(accumulator), &ptr, sizeof(void *)); }
__host__ void setStride(uint s) { hipMemcpyToSymbol(HIP_SYMBOL(stride), &s, sizeof(void *)); }
__host__ void setPathStates(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(pathStates), &ptr, sizeof(void *)); }
__host__ void setPathOrigins(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(pathOrigins), &ptr, sizeof(void *)); }
__host__ void setPathDirections(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(pathDirections), &ptr, sizeof(void *)); }
__host__ void setPathThroughputs(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(pathThroughputs), &ptr, sizeof(void *)); }
__host__ void setPotentialContributions(PotentialContribution *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(connectData), &ptr, sizeof(void *)); }
__host__ void setMaterials(DeviceMaterial *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(materials), &ptr, sizeof(void *)); }
__host__ void setFloatTextures(glm::vec4 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(floatTextures), &ptr, sizeof(void *)); }
__host__ void setUintTextures(uint *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(uintTextures), &ptr, sizeof(void *)); }
__host__ void setSkybox(glm::vec3 *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(skybox), &ptr, sizeof(void *)); }
__host__ void setSkyDimensions(uint width, uint height)
{
	hipMemcpyToSymbol(HIP_SYMBOL(skyboxWidth), &width, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(skyboxHeight), &height, sizeof(uint));
}
__host__ void setInstanceDescriptors(DeviceInstanceDescriptor *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(instances), &ptr, sizeof(void *)); }
__host__ void setGeometryEpsilon(float value) { hipMemcpyToSymbol(HIP_SYMBOL(geometryEpsilon), &value, sizeof(float)); }
__host__ void setBlueNoiseBuffer(uint *ptr) { hipMemcpyToSymbol(HIP_SYMBOL(blueNoise), &ptr, sizeof(void *)); }
__host__ void setScreenDimensions(uint width, uint height)
{
	hipMemcpyToSymbol(HIP_SYMBOL(scrWidth), &width, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(scrHeight), &height, sizeof(uint));
}
__host__ void setLightCount(rfw::LightCount lightCount) { hipMemcpyToSymbol(HIP_SYMBOL(lightCounts), &lightCount, sizeof(rfw::LightCount)); }

__host__ void setAreaLights(rfw::DeviceAreaLight *als) { hipMemcpyToSymbol(HIP_SYMBOL(areaLights), &als, sizeof(void *)); }
__host__ void setPointLights(rfw::DevicePointLight *pls) { hipMemcpyToSymbol(HIP_SYMBOL(pointLights), &pls, sizeof(void *)); }
__host__ void setSpotLights(rfw::DeviceSpotLight *sls) { hipMemcpyToSymbol(HIP_SYMBOL(spotLights), &sls, sizeof(void *)); }
__host__ void setDirectionalLights(rfw::DeviceDirectionalLight *dls) { hipMemcpyToSymbol(HIP_SYMBOL(directionalLights), &dls, sizeof(void *)); }
__host__ void setClampValue(float value) { hipMemcpyToSymbol(HIP_SYMBOL(clampValue), &value, sizeof(float)); }

__host__ const surfaceReference *getOutputSurfaceReference()
{
	const surfaceReference *ref;
	cudaGetSurfaceReference(&ref, &output);
	return ref;
}

__global__ void initCountersExtent(uint pathCount, uint sampleIndex)
{
	if (threadIdx.x != 0)
		return; // Only run a single thread
	counters->activePaths = pathCount;
	counters->shaded = 0;		 // Thread atomic for shade kernel
	counters->extensionRays = 0; // Compaction counter for extension rays
	counters->shadowRays = 0;	 // Compaction counter for connections
	counters->totalExtensionRays = pathCount;
	counters->totalShadowRays = 0;
	counters->sampleIndex = sampleIndex;
}

__global__ void initCountersSubsequent()
{
	if (threadIdx.x != 0)
		return;
	counters->totalExtensionRays += counters->extensionRays;
	counters->activePaths = counters->extensionRays; // Remaining active paths
	counters->shaded = 0;							 // Thread atomic for shade kernel
	counters->extensionRays = 0;					 // Compaction counter for extension rays
	counters->shadowRays = 0;
}

__host__ void InitCountersForExtend(unsigned int pathCount, uint sampleIndex) { initCountersExtent<<<1, 32>>>(pathCount, sampleIndex); }
__host__ void InitCountersSubsequent() { initCountersSubsequent<<<1, 32>>>(); }

__global__ void blit_buffer(const uint scrwidth, const uint scrheight, const float scale)
{
	const int x = threadIdx.x + blockIdx.x * blockDim.x;
	const int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= scrwidth || y >= scrheight)
		return;

	const auto index = x + y * scrwidth;
	const glm::vec4 value = accumulator[index] * scale;
	surf2Dwrite<glm::vec4>(value, output, x * sizeof(float4), y, hipBoundaryModeClamp);
}

__host__ hipError_t blitBuffer(const unsigned int scrwidth, const unsigned int scrheight, const uint sampleID)
{
	const unsigned int alignedWidth = NEXTMULTIPLEOF(scrwidth, 16) / 16;
	const unsigned int alignedHeight = NEXTMULTIPLEOF(scrheight, 16) / 16;
	const dim3 gridDim = dim3(alignedWidth, alignedHeight, 1);
	const dim3 blockDim = dim3(16, 16, 1);

	blit_buffer<<<gridDim, blockDim>>>(scrwidth, scrheight, 1.0f / float(sampleID));
	return hipGetLastError();
}

__device__ inline float blueNoiseSampler(int x, int y, int sampleIdx, int sampleDimension)
{
	// wrap arguments
	x &= 127;
	y &= 127;
	sampleIdx &= 255;
	sampleDimension &= 255;

	// xor index based on optimized ranking
	const int rankedSampleIndex = sampleIdx ^ blueNoise[sampleDimension + (x + y * 128) * 8 + 65536 * 3];

	// fetch value in sequence
	int value = blueNoise[sampleDimension + rankedSampleIndex * 256];

	// if the dimension is optimized, xor sequence value based on optimized scrambling
	value ^= blueNoise[(sampleDimension & 7) + (x + y * 128) * 8 + 65536];

	// convert to float and return
	return (0.5f + value) * (1.0f / 256.0f);
}

__device__ bool intersect_scene(const vec3 origin, const vec3 direction, int *instID, int *primID, float *t, vec2 *barycentrics, float t_min = 1e-5f)
{
#if !USE_TOP_MBVH
	return intersect_bvh(origin, direction, t_min, t, instID, topLevelBVH, topPrimIndices, [&](uint instance) {
		const vec3 new_origin = inverse_transforms[instance] * vec4(origin, 1);
		const vec3 new_direction = inverse_transforms[instance] * vec4(direction, 0);
		const uvec3 *indices = meshIndices[instance];
		const vec4 *vertices = meshVertices[instance];
		const uint *primIndices = meshPrimIndices[instance];

		if (indices != nullptr) // Mesh with indices
		{
#if !USE_MBVH
			return intersect_bvh(new_origin, new_direction, t_min, t, primID, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
			return intersect_mbvh(new_origin, new_direction, t_min, t, primID, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
				const uvec3 idx = indices[triangleID];
				if (intersect_triangle(new_origin, new_direction, t_min, t, vertices[idx.x], vertices[idx.y], vertices[idx.z]))
				{
					const vec3 N = normalize(cross(vec3(vertices[idx.y]) - vec3(vertices[idx.x]), vec3(vertices[idx.z]) - vec3(vertices[idx.x])));
					(*barycentrics) = get_barycentrics(new_origin + (*t) * new_direction, N, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
					return true;
				}
				return false;
			});
		}

		// Intersect mesh without indices
#if !USE_MBVH
		return intersect_bvh(new_origin, new_direction, t_min, t, primID, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
		return intersect_mbvh(new_origin, new_direction, t_min, t, primID, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
			const uvec3 idx = uvec3(triangleID * 3) + uvec3(0, 1, 2);
			if (intersect_triangle(new_origin, new_direction, t_min, t, vertices[idx.x], vertices[idx.y], vertices[idx.z]))
			{
				const vec3 N = normalize(cross(vec3(vertices[idx.y]) - vec3(vertices[idx.x]), vec3(vertices[idx.z]) - vec3(vertices[idx.x])));
				(*barycentrics) = get_barycentrics(new_origin + (*t) * new_direction, N, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
				return true;
			}
			return false;
		});
	});
#else
	return intersect_mbvh(origin, direction, t_min, t, instID, topLevelMBVH, topPrimIndices, [&](uint instance) {
		const vec3 new_origin = inverse_transforms[instance] * vec4(origin, 1);
		const vec3 new_direction = inverse_transforms[instance] * vec4(direction, 0);
		const uvec3 *indices = meshIndices[instance];
		const vec4 *vertices = meshVertices[instance];
		const uint *primIndices = meshPrimIndices[instance];

		if (indices != nullptr) // Mesh with indices
		{
#if !USE_MBVH
			return intersect_bvh(new_origin, new_direction, t_min, t, primID, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
			return intersect_mbvh(new_origin, new_direction, t_min, t, primID, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
				const uvec3 idx = indices[triangleID];
				if (intersect_triangle(new_origin, new_direction, t_min, t, vertices[idx.x], vertices[idx.y], vertices[idx.z]))
				{
					const vec3 N = normalize(cross(vec3(vertices[idx.y]) - vec3(vertices[idx.x]), vec3(vertices[idx.z]) - vec3(vertices[idx.x])));
					(*barycentrics) = get_barycentrics(new_origin + (*t) * new_direction, N, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
					return true;
				}
				return false;
			});
		}

		// Intersect mesh without indices
#if !USE_MBVH
		return intersect_bvh(new_origin, new_direction, t_min, t, primID, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
		return intersect_mbvh(new_origin, new_direction, t_min, t, primID, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
			const uvec3 idx = uvec3(triangleID * 3) + uvec3(0, 1, 2);
			if (intersect_triangle(new_origin, new_direction, t_min, t, vertices[idx.x], vertices[idx.y], vertices[idx.z]))
			{
				const vec3 N = normalize(cross(vec3(vertices[idx.y]) - vec3(vertices[idx.x]), vec3(vertices[idx.z]) - vec3(vertices[idx.x])));
				(*barycentrics) = get_barycentrics(new_origin + (*t) * new_direction, N, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
				return true;
			}
			return false;
		});
	});
#endif
}

__device__ bool is_occluded(const vec3 origin, const vec3 direction, float t_min, float t_max)
{
#if !USE_TOP_MBVH
	return intersect_bvh_shadow(origin, direction, t_min, t_max, topLevelBVH, topPrimIndices, [&](uint instance) {
		const vec3 new_origin = inverse_transforms[instance] * vec4(origin, 1);
		const vec3 new_direction = inverse_transforms[instance] * vec4(direction, 0);
		const uvec3 *indices = meshIndices[instance];
		const vec4 *vertices = meshVertices[instance];
		const uint *primIndices = meshPrimIndices[instance];

		if (indices != nullptr) // Mesh with indices
		{
#if !USE_MBVH
			return intersect_bvh_shadow(new_origin, new_direction, t_min, t_max, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
			return intersect_mbvh_shadow(new_origin, new_direction, t_min, t_max, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
				const uvec3 idx = indices[triangleID];
				return intersect_triangle(new_origin, new_direction, t_min, &t_max, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
			});
		}

		// Intersect mesh without indices
#if !USE_MBVH
		return intersect_bvh_shadow(new_origin, new_direction, t_min, t_max, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
		return intersect_mbvh_shadow(new_origin, new_direction, t_min, t_max, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
			const uvec3 idx = uvec3(triangleID * 3) + uvec3(0, 1, 2);
			return intersect_triangle(new_origin, new_direction, t_min, &t_max, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
		});
	});
#else
	return intersect_mbvh_shadow(origin, direction, t_min, t_max, topLevelMBVH, topPrimIndices, [&](uint instance) {
		const vec3 new_origin = inverse_transforms[instance] * vec4(origin, 1);
		const vec3 new_direction = inverse_transforms[instance] * vec4(direction, 0);
		const uvec3 *indices = meshIndices[instance];
		const vec4 *vertices = meshVertices[instance];

		const rfw::bvh::BVHNode *bvh = meshBVHs[instance];
		const uint *primIndices = meshPrimIndices[instance];

		if (indices != nullptr) // Mesh with indices
		{
#if !USE_MBVH
			return intersect_bvh_shadow(new_origin, new_direction, t_min, t_max, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
			return intersect_mbvh_shadow(new_origin, new_direction, t_min, t_max, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
				const uvec3 idx = indices[triangleID];
				return intersect_triangle(new_origin, new_direction, t_min, &t_max, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
			});
		}

		// Intersect mesh without indices
#if !USE_MBVH
		return intersect_bvh_shadow(new_origin, new_direction, t_min, t_max, meshBVHs[instance], primIndices, [&](uint triangleID) {
#else
		return intersect_mbvh_shadow(new_origin, new_direction, t_min, t_max, meshMBVHs[instance], primIndices, [&](uint triangleID) {
#endif
			const uvec3 idx = uvec3(triangleID * 3) + uvec3(0, 1, 2);
			return intersect_triangle(new_origin, new_direction, t_min, &t_max, vertices[idx.x], vertices[idx.y], vertices[idx.z]);
		});
	});
#endif
}

__device__ void generatePrimaryRay(const uint pathID)
{
	uint seed = WangHash(pathID * 16789 + counters->sampleIndex * 1791);

	const int sx = pathID % scrWidth;
	const int sy = pathID / scrWidth;

#if 1
	const float r0 = blueNoiseSampler(sx, sy, int(counters->sampleIndex), 0);
	const float r1 = blueNoiseSampler(sx, sy, int(counters->sampleIndex), 1);
	float r2 = blueNoiseSampler(sx, sy, int(counters->sampleIndex), 2);
	float r3 = blueNoiseSampler(sx, sy, int(counters->sampleIndex), 3);
#else
	const float r0 = RandomFloat(seed);
	const float r1 = RandomFloat(seed);
	float r2 = RandomFloat(seed);
	float r3 = RandomFloat(seed);
#endif
	const float blade = static_cast<int>(r0 * 9);
	r2 = (r2 - blade * (1.0f / 9.0f)) * 9.0f;
	float x1, y1, x2, y2;
	constexpr float piOver4point5 = 3.14159265359f / 4.5f;

	__sincosf(blade * piOver4point5, &x1, &y1);
	__sincosf((blade + 1.0f) * piOver4point5, &x2, &y2);
	if ((r2 + r3) > 1.0f)
	{
		r2 = 1.0f - r2;
		r3 = 1.0f - r3;
	}
	const float xr = x1 * r2 + x2 * r3;
	const float yr = y1 * r2 + y2 * r3;

	// TODO: Calculate this on cpu
	const vec3 right = view->p2 - view->p1;
	const vec3 up = view->p3 - view->p1;

	const vec3 origin = view->pos + view->aperture * (right * xr + up * yr);

	const float u = (static_cast<float>(sx) + r0) * (1.0f / scrWidth);
	const float v = (static_cast<float>(sy) + r1) * (1.0f / scrHeight);
	const vec3 pointOnPixel = view->p1 + u * right + v * up;
	const vec3 direction = normalize(pointOnPixel - origin);

	pathOrigins[pathID] = vec4(origin, __uint_as_float((pathID << 8) + 1 /* 1 == specular */));
	pathDirections[pathID] = vec4(direction, 0);
}

__global__ void generate_rays(uint count)
{
	const uint pathID = threadIdx.x + blockIdx.x * blockDim.x;
	if (pathID >= count)
		return;
	generatePrimaryRay(pathID);
}

__global__ void intersect_rays(IntersectionStage stage, const uint pathLength, uint count)
{
	const uint pathID = threadIdx.x + blockIdx.x * blockDim.x;
	if (pathID >= count)
		return;

	if (stage == Primary || stage == Secondary)
	{
		const uint bufferIndex = pathLength % 2;
		const uint bufferID = pathID + bufferIndex * stride;

		float t = 1e34f;
		int instID;
		int primID;
		vec2 bary;

		const vec4 O4 = pathOrigins[bufferID];
		const vec4 D4 = pathDirections[bufferID];
		const vec3 O = O4;
		const vec3 D = D4;

		vec4 result = vec4(0, 0, __int_as_float(-1), 0);
		if (intersect_scene(O, D, &instID, &primID, &t, &bary))
		{
			const vec3 p = O + t * D;
			const DeviceInstanceDescriptor &desc = instances[instID];
			const rfw::DeviceTriangle &triangle = desc.triangles[primID];
			const vec3 N = normalize(desc.invTransform * vec3(triangle.vN0.w, triangle.vN1.w, triangle.vN2.w));
			const uint barycentrics = uint(65535.0f * bary.x) + (uint(65535.0f * bary.y) << 16);
			result = vec4(__uint_as_float(barycentrics), __int_as_float(uint(instID)), __int_as_float(primID), t);
		}

		pathStates[bufferID] = result;
	}
	else if (stage == Shadow)
	{
		const vec4 O4 = connectData[pathID].Origin;
		const vec4 D4 = connectData[pathID].Direction;

		const vec3 O = vec3(O4);
		const vec3 D = vec3(D4);

		if (is_occluded(O, D, 1e-5f, D4.w - 2.0f * geometryEpsilon))
			return;

		const vec4 contribution = connectData[pathID].Emission;
		const uint pixelID = __float_as_uint(contribution.w);
		accumulator[pixelID] += vec4(vec3(contribution), 1.0f);
	}
}

__global__ __launch_bounds__(128 /* Max block size */, 4 /* Min blocks per sm */) void shade_rays(const uint pathLength, uint count)
{
	const uint jobID = threadIdx.x + blockIdx.x * blockDim.x;
	if (jobID >= count)
		return;

	const uint bufferIndex = pathLength % 2;
	const uint nextBufferIndex = 1 - bufferIndex;
	const uint bufferID = jobID + bufferIndex * stride;

	const vec4 hitData = pathStates[bufferID];
	const int primID = __float_as_int(hitData.z);

	const vec4 O4 = pathOrigins[bufferID];
	const vec4 D4 = pathDirections[bufferID];
	const uint pathID = __float_as_uint(O4.w) >> 8;
	const vec4 T4 = pathLength == 0 ? vec4(1.0f) : pathThroughputs[jobID + bufferIndex * stride];
	vec3 throughput = vec3(T4);

	if (primID < 0)
	{
		// formulas by Paul Debevec, http://www.pauldebevec.com/Probes
		const uint sky_u = static_cast<uint>(static_cast<float>(skyboxWidth - 1) * 0.5f * (1.0f + atan2(D4.x, -D4.z) * glm::one_over_pi<float>()));
		const uint sky_v = static_cast<uint>(static_cast<float>(skyboxHeight - 1) * acos(D4.y) * glm::one_over_pi<float>());
		const uint sky_idx = sky_u + sky_v * skyboxWidth;
		const vec3 skySample = skybox[sky_idx];
		vec3 contribution = throughput * vec3(skySample);

		if (any(isnan(throughput)))
			return;

		clampIntensity(contribution, clampValue);
		accumulator[pathID] += vec4(contribution, 0.0f);

		return;
	}

	const float bsdfPdf = T4.w;
	uint flags = __float_as_uint(O4.w) & 0xFF;

	const vec3 O = vec3(O4);
	const vec3 D = vec3(D4);
	const vec3 I = O + D * hitData.w;

	const uint uintBaryCentrics = __float_as_uint(hitData.x);
	const vec2 barycentrics = vec2(static_cast<float>(uintBaryCentrics & 65535), static_cast<float>(uintBaryCentrics >> 16)) * (1.0f / 65536.0f);
	const int instanceID = __float_as_int(hitData.y);
	const DeviceInstanceDescriptor &instance = instances[instanceID];
	const DeviceTriangle &triangle = instance.triangles[primID];

	vec3 N, iN, T, B;
	const ShadingData shadingData =
		getShadingData(D, barycentrics.x, barycentrics.y, view->spreadAngle * hitData.w, triangle, instanceID, N, iN, T, B, instance.invTransform);

	if (pathLength == 0 && pathID == counters->probeIdx)
	{
		counters->probedInstanceId = instanceID;
		counters->probedPrimId = primID;
		counters->probedDistance = hitData.w;
	}

	// Detect alpha in the shading code.
	if (shadingData.flags & 1)
	{
		if (pathLength < MAX_PATH_LENGTH)
		{
			if (any(isnan(throughput)))
				return;
			const uint extensionRayIdx = atomicAdd(&counters->extensionRays, 1);
			pathOrigins[extensionRayIdx + nextBufferIndex * stride] = vec4(I + D * geometryEpsilon, O4.w);
			pathDirections[extensionRayIdx + nextBufferIndex * stride] = D4;
			pathStates[extensionRayIdx + nextBufferIndex * stride] = T4;
			// TODO: this never gets hit, fix this
		}
		return;
	}

	// Terminate path on light
	if (shadingData.isEmissive()) /* r, g or b exceeds 1 */
	{
		const float DdotNL = -dot(D, N);
		vec3 contribution = vec3(0);
		if (DdotNL > 0)
		{
			if (pathLength == 0)
			{
				// Only camera rays will be treated special
				contribution = shadingData.color;
			}
			else if (flags & IS_SPECULAR)
			{
				contribution = throughput * shadingData.color;
			}
			else
			{
				// Last vertex was not specular: apply MIS
				const vec3 lastN = UnpackNormal(floatBitsToUint(D4.w));
				const float lightPdf = CalculateLightPDF(D, hitData.w, triangle.getArea(), N);
				const int triangleIdx = int(triangle.getLightTriangleIndex());
				const float pickProb = LightPickProb(triangleIdx, O, lastN, I);
				if ((bsdfPdf + lightPdf * pickProb) <= 0)
					return;

				contribution = throughput * shadingData.color * (1.0f / (bsdfPdf + lightPdf * pickProb));
			}
		}

		if (any(isnan(contribution)))
			return;

		clampIntensity(contribution, clampValue);
		accumulator[pathID] += vec4(contribution, 0.0f);
		return;
	}

	if (shadingData.getRoughness() < MIN_ROUGHNESS)
		flags |= IS_SPECULAR; // Object was specular
	else
		flags &= ~IS_SPECULAR; // Object was not specular

	uint seed = WangHash(pathID * 16789 + counters->samplesTaken * 1791 + pathLength * 720898027);
	const float flip = (dot(D, N) > 0) ? -1.0f : 1.0f;
	N *= flip;					  // Fix geometric normal
	iN *= flip;					  // Fix interpolated normal (consistent normal interpolation)
	throughput *= 1.0f / bsdfPdf; // Apply postponed bsdf pdf

	// Only cast shadow rays for non-specular objects
	if ((flags & IS_SPECULAR) == 0 &&
		(lightCounts.areaLightCount > 0 || lightCounts.pointLightCount > 0 || lightCounts.directionalLightCount > 0 || lightCounts.spotLightCount > 0))
	{
		vec3 lightColor;
		float r0, r1, pickProb, lightPdf = 0;
#if BLUENOISE
		if (counters->samplesTaken < 256)
		{
			const int x = int(pathID % scrWidth);
			const int y = int(pathID / scrWidth);
			r0 = blueNoiseSampler(blueNoise, x, y, int(counters->samplesTaken), 4);
			r1 = blueNoiseSampler(blueNoise, x, y, int(counters->samplesTaken), 5);
		}
		else
		{
			r0 = RandomFloat(seed);
			r1 = RandomFloat(seed);
		}
#else
		r0 = RandomFloat(seed);
		r1 = RandomFloat(seed);
#endif

		vec3 L = RandomPointOnLight(r0, r1, I, iN, pickProb, lightPdf, lightColor) - I;
		const float dist = length(L);
		L *= 1.0f / dist;
		const float NdotL = dot(L, iN);
		if (NdotL > 0 && lightPdf > 0)
		{
			float shadowPdf;
			const vec3 sampledBSDF = EvaluateBSDF(shadingData, iN, T, B, D * -1.0f, L, shadowPdf, seed);
			if (shadowPdf > 0)
			{
				// calculate potential contribution
				vec3 contribution = throughput * sampledBSDF * lightColor * (NdotL / (shadowPdf + lightPdf * pickProb));
				clampIntensity(contribution, clampValue);

				if (!any(isnan(contribution)))
				{
					// Add fire-and-forget shadow ray to the connections buffer
					const uint shadowRayIdx = atomicAdd(&counters->shadowRays, 1); // compaction

					connectData[shadowRayIdx].Origin = vec4(SafeOrigin(I, L, N, geometryEpsilon), 0);
					connectData[shadowRayIdx].Direction = vec4(L, dist);
					connectData[shadowRayIdx].Emission = vec4(contribution, uintBitsToFloat(pathID));
				}
			}
		}
	}

	if (pathLength >= MAX_PATH_LENGTH) // Early out in case we reached maximum path length
		return;

	vec3 R;
	float newBsdfPdf;
	vec3 bsdf = SampleBSDF(shadingData, iN, N, T, B, D * -1.0f, hitData.w, flip < 0, R, newBsdfPdf, seed);
	throughput = throughput * 1.0f / SurvivalProbability(throughput) * bsdf * abs(dot(iN, R));

#if ALLOW_DENOISER
	if (pathLength == 0)
	{
		if (counters->samplesTaken == 0)
		{
			albedos[pathIndex] = vec4(shadingData.color * abs(dot(iN, R)), 0.0f);
			normals[pathIndex] = vec4(toEyeSpace * iN, 0.0f);
		}
		else
		{
			albedos[pathIndex] += vec4(shadingData.color * abs(dot(iN, R)), 0.0f);
			normals[pathIndex] += vec4(toEyeSpace * iN, 0.0f);
		}
	}
#endif

	if (newBsdfPdf < 1e-6f || isnan(newBsdfPdf) || any(lessThan(throughput, vec3(0.0f))))
		return; // Early out in case we have an invalid bsdf

	const uint extensionRayIdx = atomicAdd(&counters->extensionRays, 1u); // Get compacted index for extension ray

	pathOrigins[extensionRayIdx + nextBufferIndex * stride] = vec4(SafeOrigin(I, R, N, geometryEpsilon), uintBitsToFloat((pathID << 8u) | flags));
	pathDirections[extensionRayIdx + nextBufferIndex * stride] = vec4(R, uintBitsToFloat(PackNormal(iN)));
	pathThroughputs[extensionRayIdx + nextBufferIndex * stride] = vec4(throughput, newBsdfPdf);
}

__host__ hipError_t generateRays(const uint count)
{
	const dim3 gridDim = dim3(NEXTMULTIPLEOF(count, 64) / 64);
	const dim3 blockDim = dim3(64);

	generate_rays<<<gridDim, blockDim>>>(count);
	return hipGetLastError();
}

__host__ hipError_t intersectRays(IntersectionStage stage, const uint pathLength, const uint count)
{
	const dim3 gridDim = dim3(NEXTMULTIPLEOF(count, 64) / 64);
	const dim3 blockDim = dim3(64);

	intersect_rays<<<gridDim, blockDim>>>(stage, pathLength, count);
	return hipGetLastError();
}

__host__ hipError_t shadeRays(const uint pathLength, const uint count)
{
	const dim3 gridDim = dim3(NEXTMULTIPLEOF(count, 64) / 64);
	const dim3 blockDim = dim3(64);

	shade_rays<<<gridDim, blockDim>>>(pathLength, count);
	return hipGetLastError();
}